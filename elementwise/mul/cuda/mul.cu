#include <hip/hip_runtime.h>

__global__
void mul_forward(const float* a, const float* b, float* out, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        out[idx] = a[idx] * b[idx];
    }
}

__global__
void mul_backward(const float* grad_out, const float* a, const float* b, float* grad_a, float* grad_b, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        grad_a[idx] = grad_out[idx] * b[idx];
        grad_b[idx] = grad_out[idx] * a[idx];
    }
}

extern "C"
void launch_mul_forward(const float* a, const float* b, float* out, int size)
{
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    mul_forward<<<blocks, threads>>>(a, b, out, size);
}

extern "C"
void launch_mul_backward(const float* grad_out, const float* a, const float* b, float* grad_a, float* grad_b, int size)
{
    int threads = 256;
    int blocks = (size + threads - 1) / threads;
    mul_backward<<<blocks, threads>>>(grad_out, a, b, grad_a, grad_b, size);
}