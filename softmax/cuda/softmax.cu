#include <hip/hip_runtime.h>

__global__
void softmax(const float *input, float *output, const int nrows, const int ncols) {

    int row_idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (row_idx < nrows) {
        int start_idx = row_idx * ncols;
        float row_sum = 0;
        float row_max = -INFINITY;

        for (int i = start_idx; i < start_idx + ncols; i ++) {
            if (input[i] > row_max)
                row_max = input[i];
        }
        
        for (int i = start_idx; i < start_idx + ncols; i ++) {
            output[i] = expf(input[i] - row_max);
            row_sum += output[i];
        }
        
        for (int i = start_idx; i < start_idx + ncols; i ++) {
            output[i] = (output[i] / row_sum);
        }
    }
}

extern "C"
void launch_softmax_forward(const float *input, float *output, const int nrows, const int ncols) {

    int threads = 256;
    int blocks = (nrows + threads - 1) / threads;
    softmax<<<blocks, threads>>>(input, output, nrows, ncols);
}